#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

#include <hipfft/hipfft.h>
#include <stdio.h>

#define N 8

#define PI 3.14159265359

#define SQR(A) ((A)*(A))
#define P3M_BRILLOUIN 1

using namespace std;

typedef struct {
  int n;
  double *pos;
  double *q;
  double *f_x;
  double *f_y;
  double *f_z;
  double alpha;
  int cao;
  int mesh;
  double box;
} data_t;

data_t *read_reference( char *filename ) {
  ifstream f;
  int i=0;
  data_t *d = (data_t *)malloc(sizeof(data_t));

  f.open(filename);

  f >> d->n;
  f >> d->cao;
  f >> d->mesh;
  f >> d->alpha;
  f >> d->box;

  d->pos = (double *)malloc(3*d->n*sizeof(double));
  d->q = (double *)malloc(d->n*sizeof(double));
  d->f_x = (double *)malloc(d->n*sizeof(double));
  d->f_y = (double *)malloc(d->n*sizeof(double));
  d->f_z = (double *)malloc(d->n*sizeof(double));

  while(f.good()) {
    f >> d->pos[3*i + 0];
    f >> d->pos[3*i + 1];
    f >> d->pos[3*i + 2];
    f >> d->q[i];
    f >> d->f_x[i];
    f >> d->f_y[i];
    f >> d->f_z[i];
    i++;
  }
  if(i != d->n)
    printf("Warning, not enought particles in file. (%d of %d)\n", i, d->n);

  return d;
}


__device__ __host__ inline double sinc(double d)
{
  double PId = PI*d;
  return (d == 0.0) ? 1.0 : sin(PId)/PId;
}

void Aliasing_sums_ik ( int cao, double box, double alpha, int mesh, int NX, int NY, int NZ,
                        double *Zaehler, double *Nenner ) {
    double S1,S2,S3;
    double fak1,fak2,zwi;
    int    MX,MY,MZ;
    double NMX,NMY,NMZ;
    double NM2;
    double expo, TE;
    double Leni = 1.0/box;

    fak1 = 1.0/ ( double ) mesh;
    fak2 = SQR ( PI/ ( alpha ) );

    Zaehler[0] = Zaehler[1] = Zaehler[2] = *Nenner = 0.0;

    for ( MX = -P3M_BRILLOUIN; MX <= P3M_BRILLOUIN; MX++ ) {
      NMX = ( ( NX > mesh/2 ) ? NX - mesh : NX ) + mesh*MX;
      S1 = pow ( sinc(fak1*NMX ), 2*cao );
      for ( MY = -P3M_BRILLOUIN; MY <= P3M_BRILLOUIN; MY++ ) {
	NMY = ( ( NY > mesh/2 ) ? NY - mesh : NY ) + mesh*MY;
	S2   = S1*pow ( sinc (fak1*NMY ), 2*cao );
	for ( MZ = -P3M_BRILLOUIN; MZ <= P3M_BRILLOUIN; MZ++ ) {
	  NMZ = ( ( NZ > mesh/2 ) ? NZ - mesh : NZ ) + mesh*MZ;
	  S3   = S2*pow ( sinc( fak1*NMZ ), 2*cao );

	  NM2 = SQR ( NMX*Leni ) + SQR ( NMY*Leni ) + SQR ( NMZ*Leni );
	  *Nenner += S3;

	  expo = fak2*NM2;
	  TE = exp ( -expo );
	  zwi  = S3 * TE/NM2;
	  Zaehler[0] += NMX*zwi*Leni;
	  Zaehler[1] += NMY*zwi*Leni;
	  Zaehler[2] += NMZ*zwi*Leni;
	}
      }
    }
}

/* Calculate influence function */
void Influence_function_berechnen_ik ( int cao, int mesh, double box, double alpha, double *G_hat ) {

  int    NX,NY,NZ;
  double Dnx,Dny,Dnz;
  double Zaehler[3]={0.0,0.0,0.0},Nenner=0.0;
  double zwi;
  int ind = 0;
  double Leni = 1.0/box;

  for ( NX=0; NX<mesh; NX++ ) {
    for ( NY=0; NY<mesh; NY++ ) {
      for ( NZ=0; NZ<mesh; NZ++ ) {
	ind = NX*mesh*mesh + NY * mesh + NZ;
	  
	if ( ( NX==0 ) && ( NY==0 ) && ( NZ==0 ) )
	  G_hat[ind]=0.0;
	else if ( ( NX% ( mesh/2 ) == 0 ) && ( NY% ( mesh/2 ) == 0 ) && ( NZ% ( mesh/2 ) == 0 ) )
	  G_hat[ind]=0.0;
	else {
	  Aliasing_sums_ik ( cao, box, alpha, mesh, NX, NY, NZ, Zaehler, &Nenner );
		  
	  Dnx = ( NX > mesh/2 ) ? NX - mesh : NX;
	  Dny = ( NY > mesh/2 ) ? NY - mesh : NY;
	  Dnz = ( NZ > mesh/2 ) ? NZ - mesh : NZ;
	    
	  zwi  = Dnx*Zaehler[0]*Leni + Dny*Zaehler[1]*Leni + Dnz*Zaehler[2]*Leni;
	  zwi /= ( ( SQR ( Dnx*Leni ) + SQR ( Dny*Leni ) + SQR ( Dnz*Leni ) ) * SQR ( Nenner ) );
	  G_hat[ind] = 2.0 * zwi / PI;
	}
      }
    }
  }
}


__device__ inline int wrap_index(const int ind, const int mesh) {
  if(ind < 0)
    return ind + mesh;
  else if(ind >= mesh)
    return ind - mesh;
  else 
    return ind;	   
}

__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__device__ float caf(int i, float x, int cao_value) {
  switch (cao_value) {
  case 1 : return 1.0;
  case 2 : {
    switch (i) {
    case 0: return 0.5-x;
    case 1: return 0.5+x;
    default:
      return 0.0;
    }
  } 
  case 3 : { 
    switch (i) {
    case 0: return 0.5*SQR(0.5 - x);
    case 1: return 0.75 - SQR(x);
    case 2: return 0.5*SQR(0.5 + x);
    default:
      return 0.0;
    }
  case 4 : { 
    switch (i) {
    case 0: return ( 1.0+x*( -6.0+x*( 12.0-x* 8.0)))/48.0;
    case 1: return (23.0+x*(-30.0+x*(-12.0+x*24.0)))/48.0;
    case 2: return (23.0+x*( 30.0+x*(-12.0-x*24.0)))/48.0;
    case 3: return ( 1.0+x*(  6.0+x*( 12.0+x* 8.0)))/48.0;
    default:
      return 0.0;
    }
  }
  case 5 : {
    switch (i) {
    case 0: return (  1.0+x*( -8.0+x*(  24.0+x*(-32.0+x*16.0))))/384.0;
    case 1: return ( 19.0+x*(-44.0+x*(  24.0+x*( 16.0-x*16.0))))/ 96.0;
    case 2: return (115.0+x*       x*(-120.0+x*       x*48.0))  /192.0;
    case 3: return ( 19.0+x*( 44.0+x*(  24.0+x*(-16.0-x*16.0))))/ 96.0;
    case 4: return (  1.0+x*(  8.0+x*(  24.0+x*( 32.0+x*16.0))))/384.0;
    default:
      return 0.0;
    }
  }
  case 6 : {
    switch (i) {
    case 0: return (  1.0+x*( -10.0+x*(  40.0+x*( -80.0+x*(  80.0-x* 32.0)))))/3840.0;
    case 1: return (237.0+x*(-750.0+x*( 840.0+x*(-240.0+x*(-240.0+x*160.0)))))/3840.0;
    case 2: return (841.0+x*(-770.0+x*(-440.0+x*( 560.0+x*(  80.0-x*160.0)))))/1920.0;
    case 3: return (841.0+x*(+770.0+x*(-440.0+x*(-560.0+x*(  80.0+x*160.0)))))/1920.0;
    case 4: return (237.0+x*( 750.0+x*( 840.0+x*( 240.0+x*(-240.0-x*160.0)))))/3840.0;
    case 5: return (  1.0+x*(  10.0+x*(  40.0+x*(  80.0+x*(  80.0+x* 32.0)))))/3840.0;
    default:
      return 0.0;
    }
  }
  case 7 : {
    switch (i) {
    case 0: return (    1.0+x*(   -12.0+x*(   60.0+x*( -160.0+x*(  240.0+x*(-192.0+x* 64.0))))))/46080.0;
    case 1: return (  361.0+x*( -1416.0+x*( 2220.0+x*(-1600.0+x*(  240.0+x*( 384.0-x*192.0))))))/23040.0;
    case 2: return (10543.0+x*(-17340.0+x*( 4740.0+x*( 6880.0+x*(-4080.0+x*(-960.0+x*960.0))))))/46080.0;
    case 3: return ( 5887.0+x*          x*(-4620.0+x*         x*( 1680.0-x*        x*320.0)))   /11520.0;
    case 4: return (10543.0+x*( 17340.0+x*( 4740.0+x*(-6880.0+x*(-4080.0+x*( 960.0+x*960.0))))))/46080.0;
    case 5: return (  361.0+x*(  1416.0+x*( 2220.0+x*( 1600.0+x*(  240.0+x*(-384.0-x*192.0))))))/23040.0;
    case 6: return (    1.0+x*(    12.0+x*(   60.0+x*(  160.0+x*(  240.0+x*( 192.0+x* 64.0))))))/46080.0;
    default:
      return 0.0;
    }
  }
  }}
  return 0.0;
}

__global__ void assign_charges(const double * const pos, const double * const q,
hipfftDoubleComplex *mesh, const int m_size, const int cao, const double pos_shift, const
double hi) {
      /** id of the particle **/
      int id = blockIdx.x;
      /** position relative to the closest gird point **/
      double m_pos[3];
      /** index of the nearest mesh point **/
      int nmp_x, nmp_y, nmp_z;      

      m_pos[0] = pos[3*id + 0] * hi - pos_shift;
      m_pos[1] = pos[3*id + 1] * hi - pos_shift;
      m_pos[2] = pos[3*id + 2] * hi - pos_shift;

      nmp_x = (int) floor(m_pos[0] + 0.5);
      nmp_y = (int) floor(m_pos[1] + 0.5);
      nmp_z = (int) floor(m_pos[2] + 0.5);

      m_pos[0] -= nmp_x;
      m_pos[1] -= nmp_y;
      m_pos[2] -= nmp_z;

      nmp_x = wrap_index(nmp_x + threadIdx.x, m_size);
      nmp_y = wrap_index(nmp_y + threadIdx.y, m_size);
      nmp_z = wrap_index(nmp_z + threadIdx.z, m_size);

      /* printf("id %d, m { %d %d %d }: weight = %lf, nmp[] = (%d %d %d), pos[] = (%lf %lf %lf)\n", id, threadIdx.x, threadIdx.y, threadIdx.z, caf(threadIdx.x, m_pos[0], cao)*caf(threadIdx.y, m_pos[1], cao)*caf(threadIdx.z, m_pos[2], cao)*q[id], nmp_x, nmp_y, nmp_z, pos[0], pos[1], pos[2]); */

      atomicAdd( &(mesh[m_size*m_size*nmp_x +  m_size*nmp_y + nmp_z].x), caf(threadIdx.x, m_pos[0], cao)*caf(threadIdx.y, m_pos[1], cao)*caf(threadIdx.z, m_pos[2], cao)*q[id]);
}

__global__ void assign_forces(const double * const pos, const double * const q,
hipfftDoubleComplex *mesh, const int m_size, const int cao, const double pos_shift, const
			      double hi, double *force, double prefactor) {
      /** id of the particle **/
      int id = blockIdx.x;
      /** position relative to the closest gird point **/
      double m_pos[3];
      /** index of the nearest mesh point **/
      int nmp_x, nmp_y, nmp_z;      

      m_pos[0] = pos[3*id + 0] * hi - pos_shift;
      m_pos[1] = pos[3*id + 1] * hi - pos_shift;
      m_pos[2] = pos[3*id + 2] * hi - pos_shift;

      nmp_x = (int) floor(m_pos[0] + 0.5);
      nmp_y = (int) floor(m_pos[1] + 0.5);
      nmp_z = (int) floor(m_pos[2] + 0.5);

      m_pos[0] -= nmp_x;
      m_pos[1] -= nmp_y;
      m_pos[2] -= nmp_z;

      nmp_x = wrap_index(nmp_x + threadIdx.x, m_size);
      nmp_y = wrap_index(nmp_y + threadIdx.y, m_size);
      nmp_z = wrap_index(nmp_z + threadIdx.z, m_size);

      /* printf("id %d, m { %d %d %d }: weight = %lf, nmp[] = (%d %d %d), pos[] = (%lf %lf %lf)\n", id, threadIdx.x, threadIdx.y, threadIdx.z, caf(threadIdx.x, m_pos[0], cao)*caf(threadIdx.y, m_pos[1], cao)*caf(threadIdx.z, m_pos[2], cao)*q[id], nmp_x, nmp_y, nmp_z, pos[0], pos[1], pos[2]); */

      atomicAdd( &(force[id]), prefactor*mesh[m_size*m_size*nmp_x +  m_size*nmp_y + nmp_z].x*caf(threadIdx.x, m_pos[0], cao)*caf(threadIdx.y, m_pos[1], cao)*caf(threadIdx.z, m_pos[2], cao)*q[id]);
}

__global__ void influence_function( double *G_hat, double box, int cao, int mesh, double alpha ) {
  int n[3];
  int linear_index;
  double nom[3] = { 0.0, 0.0, 0.0 }, dnom = 0.0;
  double fak = SQR( PI / alpha );
  int mx, my, mz;
  double box_i = 1./box;
  int nshift[3];
  int nmx, nmy, nmz;
  double zwi, nm2;
  double S1, S2, S3;

  n[0] = blockIdx.x;
  n[1] = blockIdx.y;
  n[2] = threadIdx.x;

  nshift[0] = (n[0] > mesh/2) ? n[0] - mesh : n[0];
  nshift[1] = (n[1] > mesh/2) ? n[1] - mesh : n[1];
  nshift[2] = (n[2] > mesh/2) ? n[2] - mesh : n[2];

  linear_index = SQR(mesh)*n[0] + mesh * n[1] + n[2];

  if( (n[0] == 0) && ( n[1] == 0) && n[2] == 0) {
    G_hat[linear_index] = 0.0;
    return;
  }

  if( (n[0] % (mesh/2) == 0)  && (n[1] % (mesh/2) == 0)  && (n[2] % (mesh/2) == 0)) {
    G_hat[linear_index] = 0.0;
    return;
  } 

  for ( mx = -P3M_BRILLOUIN; mx <= P3M_BRILLOUIN; mx++ ) {
    nmx = nshift[0] + mesh*mx;
    S1 = pow ( sinc ( box_i*nmx ), 2*cao );
    for ( my = -P3M_BRILLOUIN; my <= P3M_BRILLOUIN; my++ ) {
      nmy = nshift[1] + mesh*my;
      S2   = S1*pow ( sinc ( box_i*nmy ), 2*cao );
      for ( mz = -P3M_BRILLOUIN; mz <= P3M_BRILLOUIN; mz++ ) {
	nmz = nshift[2] + mesh*mz;
	S3   = S2*pow ( sinc ( box_i*nmz ), 2*cao );

	nm2 = SQR ( nmx*box_i ) + SQR ( nmy*box_i ) + SQR ( nmz*box_i );
	dnom += S3;

	zwi  = S3 * exp ( -fak*nm2 )/nm2;

	nom[0] += nmx*zwi*box_i;
	nom[1] += nmy*zwi*box_i;
	nom[2] += nmz*zwi*box_i;
      }
    }
  }
  
  zwi = box_i * (nshift[0]*nom[0] + nshift[1]*nom[1] + nshift[2]*nom[2]);
  zwi /= (SQR(nshift[0]) + SQR(nshift[1]) + SQR(nshift[2])) * SQR(box_i) *SQR(dnom);

  printf("influence_function(%d %d %d) = %lf, nm2 = %lf, nm[] = (%d %d %d), nshift[] = (%d %d %d), dnom = %e\n",
	 n[0], n[1], n[2], zwi, nm2, nmx, nmy, nmz, nshift[0], nshift[1], nshift[2], dnom);
  
  G_hat[linear_index] = 2.0 * zwi / PI;

  return;
}

__global__ void apply_influence_function( hipfftDoubleComplex *mesh, int mesh_size, double *G_hat ) {
  int linear_index = mesh_size*mesh_size*blockIdx.x + mesh_size * blockIdx.y + threadIdx.x;
  mesh[linear_index].x *= G_hat[linear_index];
  mesh[linear_index].y *= G_hat[linear_index];
}

__global__ void apply_diff_op( hipfftDoubleComplex *mesh, const int mesh_size, hipfftDoubleComplex *force_mesh,  const double box, const int dim ) {
  int linear_index = mesh_size*mesh_size*blockIdx.x + mesh_size * blockIdx.y + threadIdx.x;
  int n;

  switch( dim ) {
  case 0:
    n = blockIdx.x;
    break;
  case 1:
    n = blockIdx.y;
    break;
  case 2:
    n = threadIdx.x;
    break;
  }

  n = ( n == mesh_size/2 ) ? 0.0 : n;
  n = ( n > mesh_size/2) ? n - mesh_size : n;
 
  force_mesh[linear_index].x =  -2.0 * PI * n * mesh[linear_index].y / box;
  force_mesh[linear_index].y =   2.0 * PI * n * mesh[linear_index].x / box;
}

/* __global__ void assign_charges(const double * const pos, const double * const q, */
/* hipfftDoubleComplex *mesh, const int m_size, const int cao, const double pos_shift, const */
/* double hi) { */

int main(void) {
  hipfftHandle plan;
  hipfftDoubleComplex *data, *force_mesh;

  const double pos_h[6] = {  4.0,  5.0,  5.0, 6.0, 5.0, 5.0 }, q_h[2] = { -1.0, 1.0 };
  double *pos_d, *q_d;
  const int cao = 7;
  const double box = 10.0;

  double forces_h[3][2], *forces_d;

  data_t *d;

  double *g_hat_d, *g_hat_h;
  hipMalloc((void**)&g_hat_d, sizeof(double)*N*N*N);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    return 0;	
  }

  d = read_reference("test.dat");

  for(int i =0; i<d->n; ++i) {
    printf("pos %lf %lf %lf q %lf\n", d->pos[3*i+0], d->pos[3*i+1], d->pos[3*i+2], d->q[i]);
  }

  g_hat_h = (double *)malloc(N*N*N*sizeof(double));

  hipfftDoubleComplex *data_h = (hipfftDoubleComplex *) malloc( N * N * N * sizeof(hipfftDoubleComplex));

  hipMalloc((void**)&data, sizeof(hipfftDoubleComplex)*N*N*N);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    return 0;	
  }

  hipMalloc((void**)&force_mesh, sizeof(hipfftDoubleComplex)*N*N*N);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    return 0;	
  }

  hipMalloc((void**)&pos_d, sizeof(pos_h));
  hipMalloc((void**)&q_d, sizeof(q_h));
  hipMalloc((void**)&forces_d, sizeof(forces_h));

  hipMemcpy( pos_d, pos_h, sizeof(pos_h), hipMemcpyHostToDevice);
  hipMemcpy( q_d, q_h, sizeof(q_h), hipMemcpyHostToDevice);

  // prepare influence function
  dim3 blockDim(N, N, 1);
  dim3 thdDim( N, 1, 1);

  Influence_function_berechnen_ik( cao, N, box, 0.8, g_hat_h );

  hipMemcpy( g_hat_d, g_hat_h, N*N*N*sizeof(double), hipMemcpyHostToDevice);
\
  if (hipDeviceSynchronize() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    return 0;
  }

  hipMemset( data, 0, N*N*N*sizeof(hipfftDoubleComplex));
  
  dim3 caoBlock(cao, cao, cao);

  assign_charges<<<2, caoBlock>>>( pos_d, q_d, data, N, cao, (cao-1)/2.0, N/box);

  hipDeviceSynchronize();

  if (hipfftPlan3d(&plan, N, N, N, HIPFFT_Z2Z) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: Plan creation failed");
    return 0;
  }

  if (hipfftExecZ2Z(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed\n");
    return 0;
  }

  if (hipDeviceSynchronize() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    return 0;
  }

  apply_influence_function<<<blockDim, thdDim>>>( data, N, g_hat_d);

  for(int dim = 0; dim < 3; dim++) {
    if (hipDeviceSynchronize() != hipSuccess){
      fprintf(stderr, "Cuda error: Failed to synchronize\n");
      return 0;
    }

    apply_diff_op<<<blockDim, thdDim>>>( data, N, force_mesh, box, dim);

    if (hipDeviceSynchronize() != hipSuccess){
      fprintf(stderr, "Cuda error: Failed to synchronize diff_op\n");
      return 0;
    }

    /* Use the CUFFT plan to transform the signal in place. */
    if (hipfftExecZ2Z(plan, force_mesh, force_mesh, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
      fprintf(stderr, "CUFFT error: ExecZ2Z Backward failed\n");
      return 0;
    }

    if (hipDeviceSynchronize() != hipSuccess){
      fprintf(stderr, "Cuda error: Failed to synchronize back\n");
      return 0;
    }

    hipMemset(forces_d, 0, sizeof(forces_h));

    assign_forces<<< 2, caoBlock>>>( pos_d, q_d, force_mesh, N, cao, (cao-1)/2.0, N/box, forces_d, 1.0 / ( 2.0 *  box * box * box));

    hipMemcpy( forces_h[dim], forces_d, sizeof(forces_h), hipMemcpyDeviceToHost);
  }

  hipMemcpy( data_h, data, sizeof(hipfftDoubleComplex)*N*N*N, hipMemcpyDeviceToHost);

  hipfftDestroy(plan);
  hipFree(data);

  printf("f_x = [%lf %lf]\n", forces_h[0][0], forces_h[0][1]); 
  printf("f_y = [%lf %lf]\n", forces_h[1][0], forces_h[1][1]); 
  printf("f_z = [%lf %lf]\n", forces_h[2][0], forces_h[2][1]); 

}

