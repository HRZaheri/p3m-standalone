
#include <hip/hip_runtime.h>
#define PI 3.14159

#define P3M_BRILLOUIN 1
#define SQR(A) ((A)*(A))

__device__ double sinc(double d)
{
  double PId = PI*d;
  return (d == 0.0) ? 1.0 : sin(PId)/PId;
}

__global__ void influence_function( double *G_hat, double box, int cao, int mesh, double alpha ) {
  int n[3];
  int linear_index;
  double nom[3] = { 0.0, 0.0, 0.0 }, dnom = 0.0;
  double fak = SQR( PI / alpha );
  int mx, my, mz;
  double box_i = 1./box;
  int nshift[3];
  int nmx, nmy, nmz, nm2;
  double zwi;
  double S1, S2, S3;
  
  n[0] = blockDim.x * blockIdx.x;
  n[1] = blockDim.y * blockIdx.y;
  n[2] = threadIdx.x;

  nshift[0] = n[0] - round(n[0]/(double)mesh) * (double)mesh;
  nshift[1] = n[1] - round(n[1]/(double)mesh) * (double)mesh;
  nshift[2] = n[2] - round(n[2]/(double)mesh) * (double)mesh;

  linear_index = SQR(mesh)*n[0] + mesh * n[1] + n[2];

  if( (n[0] % (mesh/2))  && (n[1] % (mesh/2))  && (n[2] % (mesh/2))) {
    G_hat[linear_index] = 0.0;
    return;
  } 

  for ( mx = -P3M_BRILLOUIN; mx <= P3M_BRILLOUIN; mx++ ) {
    nmx = nshift[n[0]] + mesh*mx;
    S1 = pow ( sinc ( box_i*nmx ), 2*cao );
    for ( my = -P3M_BRILLOUIN; my <= P3M_BRILLOUIN; my++ ) {
      nmy = nshift[n[1]] + mesh*my;
      S2   = S1*pow ( sinc ( box_i*nmy ), 2*cao );
      for ( mz = -P3M_BRILLOUIN; mz <= P3M_BRILLOUIN; mz++ ) {
	nmz = nshift[n[2]] + mesh*mz;
	S3   = S2*pow ( sinc ( box_i*nmz ), 2*cao );

	nm2 = SQR ( nmx*box_i ) + SQR ( nmy*box_i ) + SQR ( nmz*box_i );
	dnom += S3;

	zwi  = S3 * exp ( -fak*nm2 )/nm2;
	nom[0] += nmx*zwi*box_i;
	nom[1] += nmy*zwi*box_i;
	nom[2] += nmz*zwi*box_i;
      }
    }
  }
  
  zwi = box_i * (nshift[0]*nom[0] + nshift[1]*nom[1] + nshift[2]*nom[2]);
  zwi /= (SQR(nshift[0]) + SQR(nshift[1]) + SQR(nshift[2])) * SQR(box_i) *SQR(dnom);
  
  G_hat[linear_index] = 2.0 * zwi / PI;

  return;
}

__global__ void convolute( double *G_hat, double *mesh, int mesh_size ) {
  int n[3];
  int linear_index;
  n[0] = blockDim.x * blockIdx.x;
  n[1] = blockDim.y * blockIdx.y;
  n[2] = threadIdx.x;
  
  linear_index = SQR(mesh_size)*n[0] + mesh_size * n[1] + n[2];

  mesh[linear_index] *= G_hat[linear_index];
}

