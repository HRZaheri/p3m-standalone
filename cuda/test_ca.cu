#include "assignment.h"

#include <stdlib.h>

#include <gsl/gsl_rng.h>

#include <hipfft/hipfft.h>

#define MESH_SIZE 32
#define PARTICLES 8
#define CAO 5
#define BOX 15.0

#define FLOAT_TYPE float



void generate_random_system(int size, FLOAT_TYPE box, FLOAT_TYPE max_charge, FLOAT_TYPE *pos, FLOAT_TYPE *q) {
  int i,j;
  gsl_rng *rng = gsl_rng_alloc(gsl_rng_default);

  for(i=0;i<size;i++) {
    for(j=0;j<3;j++) {
      pos[3*i+j] = box*gsl_rng_uniform(rng);
    }
    q[i] = (1.0 - 2.0 * (i%2)) * max_charge;
  }

  gsl_rng_free (rng);
}


int main(void) {

  hipfftComplex *h_mesh = (hipfftComplex *)malloc(MESH_SIZE*MESH_SIZE*MESH_SIZE*sizeof(hipfftComplex));
  FLOAT_TYPE *h_part = (FLOAT_TYPE *)malloc(3*PARTICLES*sizeof(FLOAT_TYPE));
  FLOAT_TYPE *h_q    = (FLOAT_TYPE *)malloc(PARTICLES*sizeof(FLOAT_TYPE));

  FLOAT_TYPE *d_mesh, *d_part, *d_q;

  FLOAT_TYPE h = BOX / MESH_SIZE;

  FLOAT_TYPE sum = 0.0;

  hipfftHandle plan;

  generate_random_system( PARTICLES, BOX, 1.0, h_part, h_q);

  for(int i = 0; i< PARTICLES; i++)
    printf("part %d, q = %f, pos = (%f %f %f)\n", i, h_q[i], h_part[3*i+0], h_part[3*i+1], h_part[3*i+2]);

  
  hipMalloc((void**)&d_mesh, MESH_SIZE*MESH_SIZE*MESH_SIZE*sizeof(hipfftComplex));
  hipMalloc((void**)&d_part, 3*PARTICLES*sizeof(FLOAT_TYPE));
  hipMalloc((void**)&d_q, PARTICLES*sizeof(FLOAT_TYPE));

  hipMemcpy( d_part, h_part, 3*PARTICLES*sizeof(FLOAT_TYPE), hipMemcpyHostToDevice);
  hipMemcpy( d_q, h_q, PARTICLES*sizeof(FLOAT_TYPE), hipMemcpyHostToDevice);
  hipMemset( d_mesh, 0, MESH_SIZE*MESH_SIZE*MESH_SIZE*sizeof(hipfftComplex));

  dim3 grid(PARTICLES,1), block(CAO,CAO,CAO);

  assign_charges<<<grid, block>>>( d_part, d_q, d_mesh, MESH_SIZE, CAO, (FLOAT_TYPE) ((CAO-1)/2), 1./h);

  hipMemcpy( h_mesh, d_mesh, MESH_SIZE*MESH_SIZE*MESH_SIZE*sizeof(hipfftComplex), hipMemcpyDeviceToHost);

  for(int i = 0; i< MESH_SIZE*MESH_SIZE*MESH_SIZE; i++) {
    printf("%d %f\n", i, h_mesh[i].x);
    sum += h_mesh[i].x;
  }
  printf("sum %f\n", sum);
}

