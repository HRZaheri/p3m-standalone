#include "assignment.h"

#include <stdlib.h>

#include <gsl/gsl_rng.h>

#define mesh_size 32
#define particles 512
#define cao 5
#define box 15.0

int main(void) {
  float *h_mesh = (float *)malloc(mesh_size*sizeof(float));
  float *h_part = (float *)malloc(3*particles*sizeof(float));
  float *h_q    = (float *)malloc(particles*sizeof(float));

  float *d_mesh, *d_part, *d_q;

  float h = box / mesh_size;

  for(int i = 0; i<particles; i++) {

  }

  hipMalloc((void**)&d_mesh, mesh_size*sizeof(float));
  hipMalloc((void**)&d_part, 3*particles*sizeof(float));
  hipMalloc((void**)&d_q, particles**sizeof(float));
}

