#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>

#define PI 3.14159265359

#include "p3m-ik-cuda.h"

#define SQR(A) ((A)*(A))
#define P3M_BRILLOUIN 1

using namespace std;

typedef struct {
  hipfftHandle plan;
  hipfftDoubleComplex *charge_mesh;
  hipfftDoubleComplex *force_mesh;
  double *g_hat_d;
  double *pos_d;
  double *q_d;
  double *forces_d;
  p3m_cuda_data_t *d;
} p3m_cuda_state_t;

void p3m_ik_cuda_free( p3m_cuda_data_t *d) {
  p3m_cuda_state_t *p3m_cuda_state = (p3m_cuda_state_t *) d->s;  

  hipfftDestroy(p3m_cuda_state->plan);
 
  hipFree(p3m_cuda_state->charge_mesh);
  hipFree(p3m_cuda_state->force_mesh);

  hipFree(p3m_cuda_state->g_hat_d);
  hipFree(p3m_cuda_state->q_d);
  hipFree(p3m_cuda_state->pos_d);
  hipFree(p3m_cuda_state->forces_d);
}

__device__ __host__ inline static double sinc(double d)
{
  double PId = PI*d;
  return (d == 0.0) ? 1.0 : sin(PId)/PId;
}

void static Aliasing_sums_ik ( int cao, double box, double alpha, int mesh, int NX, int NY, int NZ,
                        double *Zaehler, double *Nenner ) {
    double S1,S2,S3;
    double fak1,fak2,zwi;
    int    MX,MY,MZ;
    double NMX,NMY,NMZ;
    double NM2;
    double expo, TE;
    double Leni = 1.0/box;

    fak1 = 1.0/ ( double ) mesh;
    fak2 = SQR ( PI/ ( alpha ) );

    Zaehler[0] = Zaehler[1] = Zaehler[2] = *Nenner = 0.0;

    for ( MX = -P3M_BRILLOUIN; MX <= P3M_BRILLOUIN; MX++ ) {
      NMX = ( ( NX > mesh/2 ) ? NX - mesh : NX ) + mesh*MX;
      S1 = pow ( sinc(fak1*NMX ), 2*cao );
      for ( MY = -P3M_BRILLOUIN; MY <= P3M_BRILLOUIN; MY++ ) {
	NMY = ( ( NY > mesh/2 ) ? NY - mesh : NY ) + mesh*MY;
	S2   = S1*pow ( sinc (fak1*NMY ), 2*cao );
	for ( MZ = -P3M_BRILLOUIN; MZ <= P3M_BRILLOUIN; MZ++ ) {
	  NMZ = ( ( NZ > mesh/2 ) ? NZ - mesh : NZ ) + mesh*MZ;
	  S3   = S2*pow ( sinc( fak1*NMZ ), 2*cao );

	  NM2 = SQR ( NMX*Leni ) + SQR ( NMY*Leni ) + SQR ( NMZ*Leni );
	  *Nenner += S3;

	  expo = fak2*NM2;
	  TE = exp ( -expo );
	  zwi  = S3 * TE/NM2;
	  Zaehler[0] += NMX*zwi*Leni;
	  Zaehler[1] += NMY*zwi*Leni;
	  Zaehler[2] += NMZ*zwi*Leni;
	}
      }
    }
}

/* Calculate influence function */
void static Influence_function_berechnen_ik ( int cao, int mesh, double box, double alpha, double *G_hat ) {

  int    NX,NY,NZ;
  double Dnx,Dny,Dnz;
  double Zaehler[3]={0.0,0.0,0.0},Nenner=0.0;
  double zwi;
  int ind = 0;
  double Leni = 1.0/box;

  for ( NX=0; NX<mesh; NX++ ) {
    for ( NY=0; NY<mesh; NY++ ) {
      for ( NZ=0; NZ<mesh; NZ++ ) {
	ind = NX*mesh*mesh + NY * mesh + NZ;
	  
	if ( ( NX==0 ) && ( NY==0 ) && ( NZ==0 ) )
	  G_hat[ind]=0.0;
	else if ( ( NX% ( mesh/2 ) == 0 ) && ( NY% ( mesh/2 ) == 0 ) && ( NZ% ( mesh/2 ) == 0 ) )
	  G_hat[ind]=0.0;
	else {
	  Aliasing_sums_ik ( cao, box, alpha, mesh, NX, NY, NZ, Zaehler, &Nenner );
		  
	  Dnx = ( NX > mesh/2 ) ? NX - mesh : NX;
	  Dny = ( NY > mesh/2 ) ? NY - mesh : NY;
	  Dnz = ( NZ > mesh/2 ) ? NZ - mesh : NZ;
	    
	  zwi  = Dnx*Zaehler[0]*Leni + Dny*Zaehler[1]*Leni + Dnz*Zaehler[2]*Leni;
	  zwi /= ( ( SQR ( Dnx*Leni ) + SQR ( Dny*Leni ) + SQR ( Dnz*Leni ) ) * SQR ( Nenner ) );
	  G_hat[ind] = 2.0 * zwi / PI;
	}
      }
    }
  }
}


__device__ inline int wrap_index(const int ind, const int mesh) {
  if(ind < 0)
    return ind + mesh;
  else if(ind >= mesh)
    return ind - mesh;
  else 
    return ind;	   
}

__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__device__ double caf(int i, double x, int cao_value) {
  switch (cao_value) {
  case 1 : return 1.0;
  case 2 : {
    switch (i) {
    case 0: return 0.5-x;
    case 1: return 0.5+x;
    default:
      return 0.0;
    }
  } 
  case 3 : { 
    switch (i) {
    case 0: return 0.5*SQR(0.5 - x);
    case 1: return 0.75 - SQR(x);
    case 2: return 0.5*SQR(0.5 + x);
    default:
      return 0.0;
    }
  case 4 : { 
    switch (i) {
    case 0: return ( 1.0+x*( -6.0+x*( 12.0-x* 8.0)))/48.0;
    case 1: return (23.0+x*(-30.0+x*(-12.0+x*24.0)))/48.0;
    case 2: return (23.0+x*( 30.0+x*(-12.0-x*24.0)))/48.0;
    case 3: return ( 1.0+x*(  6.0+x*( 12.0+x* 8.0)))/48.0;
    default:
      return 0.0;
    }
  }
  case 5 : {
    switch (i) {
    case 0: return (  1.0+x*( -8.0+x*(  24.0+x*(-32.0+x*16.0))))/384.0;
    case 1: return ( 19.0+x*(-44.0+x*(  24.0+x*( 16.0-x*16.0))))/ 96.0;
    case 2: return (115.0+x*       x*(-120.0+x*       x*48.0))  /192.0;
    case 3: return ( 19.0+x*( 44.0+x*(  24.0+x*(-16.0-x*16.0))))/ 96.0;
    case 4: return (  1.0+x*(  8.0+x*(  24.0+x*( 32.0+x*16.0))))/384.0;
    default:
      return 0.0;
    }
  }
  case 6 : {
    switch (i) {
    case 0: return (  1.0+x*( -10.0+x*(  40.0+x*( -80.0+x*(  80.0-x* 32.0)))))/3840.0;
    case 1: return (237.0+x*(-750.0+x*( 840.0+x*(-240.0+x*(-240.0+x*160.0)))))/3840.0;
    case 2: return (841.0+x*(-770.0+x*(-440.0+x*( 560.0+x*(  80.0-x*160.0)))))/1920.0;
    case 3: return (841.0+x*(+770.0+x*(-440.0+x*(-560.0+x*(  80.0+x*160.0)))))/1920.0;
    case 4: return (237.0+x*( 750.0+x*( 840.0+x*( 240.0+x*(-240.0-x*160.0)))))/3840.0;
    case 5: return (  1.0+x*(  10.0+x*(  40.0+x*(  80.0+x*(  80.0+x* 32.0)))))/3840.0;
    default:
      return 0.0;
    }
  }
  case 7 : {
    switch (i) {
    case 0: return (    1.0+x*(   -12.0+x*(   60.0+x*( -160.0+x*(  240.0+x*(-192.0+x* 64.0))))))/46080.0;
    case 1: return (  361.0+x*( -1416.0+x*( 2220.0+x*(-1600.0+x*(  240.0+x*( 384.0-x*192.0))))))/23040.0;
    case 2: return (10543.0+x*(-17340.0+x*( 4740.0+x*( 6880.0+x*(-4080.0+x*(-960.0+x*960.0))))))/46080.0;
    case 3: return ( 5887.0+x*          x*(-4620.0+x*         x*( 1680.0-x*        x*320.0)))   /11520.0;
    case 4: return (10543.0+x*( 17340.0+x*( 4740.0+x*(-6880.0+x*(-4080.0+x*( 960.0+x*960.0))))))/46080.0;
    case 5: return (  361.0+x*(  1416.0+x*( 2220.0+x*( 1600.0+x*(  240.0+x*(-384.0-x*192.0))))))/23040.0;
    case 6: return (    1.0+x*(    12.0+x*(   60.0+x*(  160.0+x*(  240.0+x*( 192.0+x* 64.0))))))/46080.0;
    default:
      return 0.0;
    }
  }
  }}
  return 0.0;
}

__global__ void assign_charges(const double * const pos, const double * const q,
hipfftDoubleComplex *mesh, const int m_size, const int cao, const double pos_shift, const
double hi) {
      /** id of the particle **/
      int id = blockIdx.x;
      /** position relative to the closest gird point **/
      double m_pos[3];
      /** index of the nearest mesh point **/
      int nmp_x, nmp_y, nmp_z;      

      m_pos[0] = pos[3*id + 0] * hi - pos_shift;
      m_pos[1] = pos[3*id + 1] * hi - pos_shift;
      m_pos[2] = pos[3*id + 2] * hi - pos_shift;

      nmp_x = (int) floor(m_pos[0] + 0.5);
      nmp_y = (int) floor(m_pos[1] + 0.5);
      nmp_z = (int) floor(m_pos[2] + 0.5);

      m_pos[0] -= nmp_x;
      m_pos[1] -= nmp_y;
      m_pos[2] -= nmp_z;

      nmp_x = wrap_index(nmp_x + threadIdx.x, m_size);
      nmp_y = wrap_index(nmp_y + threadIdx.y, m_size);
      nmp_z = wrap_index(nmp_z + threadIdx.z, m_size);

      /* printf("id %d, m { %d %d %d }: weight = %lf, nmp[] = (%d %d %d), pos[] = (%lf %lf %lf)\n", id, threadIdx.x, threadIdx.y, threadIdx.z, caf(threadIdx.x, m_pos[0], cao)*caf(threadIdx.y, m_pos[1], cao)*caf(threadIdx.z, m_pos[2], cao)*q[id], nmp_x, nmp_y, nmp_z, m_pos[0], m_pos[1], m_pos[2]); */

      atomicAdd( &(mesh[m_size*m_size*nmp_x +  m_size*nmp_y + nmp_z].x), caf(threadIdx.x, m_pos[0], cao)*caf(threadIdx.y, m_pos[1], cao)*caf(threadIdx.z, m_pos[2], cao)*q[id]);
}

__global__ void assign_forces(const double * const pos, const double * const q,
hipfftDoubleComplex *mesh, const int m_size, const int cao, const double pos_shift, const
			      double hi, double *force, double prefactor) {
      /** id of the particle **/
      int id = blockIdx.x;
      /** position relative to the closest gird point **/
      double m_pos[3];
      /** index of the nearest mesh point **/
      int nmp_x, nmp_y, nmp_z;      

      m_pos[0] = pos[3*id + 0] * hi - pos_shift;
      m_pos[1] = pos[3*id + 1] * hi - pos_shift;
      m_pos[2] = pos[3*id + 2] * hi - pos_shift;

      nmp_x = (int) floor(m_pos[0] + 0.5);
      nmp_y = (int) floor(m_pos[1] + 0.5);
      nmp_z = (int) floor(m_pos[2] + 0.5);

      m_pos[0] -= nmp_x;
      m_pos[1] -= nmp_y;
      m_pos[2] -= nmp_z;

      nmp_x = wrap_index(nmp_x + threadIdx.x, m_size);
      nmp_y = wrap_index(nmp_y + threadIdx.y, m_size);
      nmp_z = wrap_index(nmp_z + threadIdx.z, m_size);

      /* printf("id %d, m { %d %d %d }: weight = %lf, nmp[] = (%d %d %d), pos[] = (%lf %lf %lf)\n", id, threadIdx.x, threadIdx.y, threadIdx.z, caf(threadIdx.x, m_pos[0], cao)*caf(threadIdx.y, m_pos[1], cao)*caf(threadIdx.z, m_pos[2], cao)*q[id], nmp_x, nmp_y, nmp_z, pos[0], pos[1], pos[2]); */

      atomicAdd( &(force[id]), -prefactor*mesh[m_size*m_size*nmp_x +  m_size*nmp_y + nmp_z].x*caf(threadIdx.x, m_pos[0], cao)*caf(threadIdx.y, m_pos[1], cao)*caf(threadIdx.z, m_pos[2], cao)*q[id]);
}

__global__ void apply_influence_function( hipfftDoubleComplex *mesh, int mesh_size, double *G_hat ) {
  int linear_index = mesh_size*mesh_size*blockIdx.x + mesh_size * blockIdx.y + threadIdx.x;
  mesh[linear_index].x *= G_hat[linear_index];
  mesh[linear_index].y *= G_hat[linear_index];
}

__global__ void apply_diff_op( hipfftDoubleComplex *mesh, const int mesh_size, hipfftDoubleComplex *force_mesh,  const double box, const int dim ) {
  int linear_index = mesh_size*mesh_size*blockIdx.x + mesh_size * blockIdx.y + threadIdx.x;
  int n;

  switch( dim ) {
  case 0:
    n = blockIdx.x;
    break;
  case 1:
    n = blockIdx.y;
    break;
  case 2:
    n = threadIdx.x;
    break;
  }

  n = ( n == mesh_size/2 ) ? 0.0 : n;
  n = ( n > mesh_size/2) ? n - mesh_size : n;
 
  force_mesh[linear_index].x =  -2.0 * PI * n * mesh[linear_index].y / box;
  force_mesh[linear_index].y =   2.0 * PI * n * mesh[linear_index].x / box;
}

/* __global__ void assign_charges(const double * const pos, const double * const q, */
/* hipfftDoubleComplex *mesh, const int m_size, const int cao, const double pos_shift, const */
/* double hi) { */
 
void p3m_ik_cuda_init( p3m_cuda_data_t *d ) {
  puts("p3m_ik_cuda_init():");
  double *g_hat_h = (double *)malloc(d->mesh*d->mesh*d->mesh*sizeof(double));
  p3m_cuda_state_t *p3m_cuda_state = (p3m_cuda_state_t *) malloc(sizeof(p3m_cuda_state_t));

  puts("Allocating g_hat_d.");
  hipMalloc((void**)&(p3m_cuda_state->g_hat_d), sizeof(double)*d->mesh*d->mesh*d->mesh);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "p3m_cuda: Failed to allocate\n");
  }

  puts("Allocating charge_mesh.");
  hipMalloc((void**)&(p3m_cuda_state->charge_mesh), sizeof(hipfftDoubleComplex)*d->mesh*d->mesh*d->mesh);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "p3m_cuda: Failed to allocate\n");
  }

  puts("Allocating foce_mesh.");
  hipMalloc((void**)&(p3m_cuda_state->force_mesh), sizeof(hipfftDoubleComplex)*d->mesh*d->mesh*d->mesh);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "p3m_cuda: Failed to allocate\n");
  }


  hipMalloc((void**)&(p3m_cuda_state->pos_d), 3*d->n*sizeof(double));
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "p3m_cuda: Failed to allocate\n");
  }
  hipMalloc((void**)&(p3m_cuda_state->q_d), d->n*sizeof(double));
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "p3m_cuda: Failed to allocate\n");
  }
  hipMalloc((void**)&(p3m_cuda_state->forces_d), d->n*sizeof(double));
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "p3m_cuda: Failed to allocate\n");
  }

  Influence_function_berechnen_ik( d->cao, d->mesh, d->box, d->alpha, g_hat_h );  

  hipMemcpy( p3m_cuda_state->g_hat_d, g_hat_h, d->mesh*d->mesh*d->mesh*sizeof(double), hipMemcpyHostToDevice);

  if (hipfftPlan3d(&(p3m_cuda_state->plan), d->mesh, d->mesh, d->mesh, HIPFFT_Z2Z) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: Plan creation failed");
  }
}

void p3m_ik_cuda(p3m_cuda_data_t *d) {
  p3m_cuda_state_t *p3m_cuda_state = (p3m_cuda_state_t *) d->s;

  hipMemcpy( p3m_cuda_state->pos_d, d->pos, 3*d->n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy( p3m_cuda_state->q_d, d->q, d->n*sizeof(double), hipMemcpyHostToDevice);

  dim3 blockDim(d->mesh, d->mesh, 1);
  dim3 thdDim( d->mesh, 1, 1);

  if (hipDeviceSynchronize() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    return;
  }

  hipMemset( p3m_cuda_state->charge_mesh, 0, d->mesh*d->mesh*d->mesh*sizeof(hipfftDoubleComplex));
  
  dim3 caoBlock(d->cao, d->cao, d->cao);

  assign_charges<<<d->n, caoBlock>>>( p3m_cuda_state->pos_d, p3m_cuda_state->q_d, p3m_cuda_state->charge_mesh, d->mesh, d->cao,(double)((d->cao-1)/2), d->mesh/d->box);

  hipDeviceSynchronize();

  if (hipfftExecZ2Z(p3m_cuda_state->plan, p3m_cuda_state->charge_mesh, p3m_cuda_state->charge_mesh, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed\n");
    return;
  }

  if (hipDeviceSynchronize() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    return;
  }

  apply_influence_function<<<blockDim, thdDim>>>( p3m_cuda_state->charge_mesh, d->mesh, p3m_cuda_state->g_hat_d);

  for(int dim = 0; dim < 3; dim++) {
    if (hipDeviceSynchronize() != hipSuccess){
      fprintf(stderr, "Cuda error: Failed to synchronize\n");
      return;
    }

    apply_diff_op<<<blockDim, thdDim>>>( p3m_cuda_state->charge_mesh, d->mesh, p3m_cuda_state->force_mesh, d->box, dim);

    if (hipDeviceSynchronize() != hipSuccess){
      fprintf(stderr, "Cuda error: Failed to synchronize diff_op\n");
      return;
    }

    /* Use the CUFFT plan to transform the signal in place. */
    if (hipfftExecZ2Z(p3m_cuda_state->plan, p3m_cuda_state->force_mesh, p3m_cuda_state->force_mesh, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
      fprintf(stderr, "CUFFT error: ExecZ2Z Backward failed\n");
      return;
    }

    if (hipDeviceSynchronize() != hipSuccess){
      fprintf(stderr, "Cuda error: Failed to synchronize back\n");
      return;
    }

    hipMemset(p3m_cuda_state->forces_d, 0, d->n*sizeof(double));

    assign_forces<<< d->n, caoBlock>>>( p3m_cuda_state->pos_d, p3m_cuda_state->q_d, p3m_cuda_state->force_mesh, d->mesh, d->cao, (double)((d->cao-1)/2), d->mesh/d->box, p3m_cuda_state->forces_d, 1.0 / ( 2.0 *  d->box * d->box * d->box));

    hipMemcpy( d->f[dim], p3m_cuda_state->forces_d, d->n*sizeof(double), hipMemcpyDeviceToHost);
  }
}

